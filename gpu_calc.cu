#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"


__global__ void device_aes_encrypt(unsigned char *pt, int *rkey, unsigned char *ct, long int size){

  //This kernel executes AES encryption on a GPU.
  //Please modify this kernel!!
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int data[16];
  memcpy(data, pt, 16);
  if(thread_id == 0)
    printf("size = %ld\n", size);

  printf("You can use printf function to eliminate bugs in your kernel.\n");
  printf("This thread ID is %d.\n", thread_id);

  //...
  PrintText(*data);
}


__device__ void SubBytes(){

}

__device__ void ShiftRows(){

}

__device__ void MixColumns(){

}

__device__ void AddRoundKey(){

}
__device__ void PrintText(int *state){
  for (int i = 0; i < 16; i++) {
    printf("%d ", state[i]);
  }
  printf("\n");
}
__device__ void Cipher(int *state, int *rkey){
  // int rnd;
  // int i;
  //
  // AddRoundKey();
  //
  // for(rnd = 1; rnd < NR; rnd++){
  //   SubBytes(state);
  //   ShiftRows(state);
  //   MixColumns(state);
  //   AddRoundKey(state, rkey, rnd);
  // }
  //
  // SubBytes(state);
  // ShiftRows(state);
  // AddRoundKey(state, rkey, rnd);
  //
  // return 0;
}

void launch_aes_kernel(unsigned char *pt, int *rk, unsigned char *ct, long int size){

  //This function launches the AES kernel.
  //Please modify this function for AES kernel.
  //In this function, you need to allocate the device memory and so on.

  unsigned char *d_pt, *d_ct;
  int *d_rkey;

  dim3 dim_grid(FILESIZE/16,1,1), dim_block(1,1,1);

  hipMalloc((void **)&d_pt, sizeof(unsigned char)*size);
  hipMalloc((void **)&d_rkey, sizeof(int)*44);
  hipMalloc((void **)&d_ct, sizeof(unsigned char)*size);

  hipMemset(d_pt, 0, sizeof(unsigned char)*size);
  hipMemcpy(d_pt, pt, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
  hipMemcpy(d_rkey, rk, sizeof(int)*44, hipMemcpyHostToDevice);

  device_aes_encrypt<<<dim_grid, dim_block>>>(d_pt, d_rkey, d_ct, size);

  hipMemcpy(ct, d_ct, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);

  hipFree(d_pt);
  hipFree(d_ct);
}
