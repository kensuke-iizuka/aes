#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

#define NB (4)
#define NBb (16)

#define NK (4)
#define NR (10)
#define FILESIZE (16*128*13*16*512)
#define STREAM_NUM (32)

__device__ void gpuSubBytes(int *,int *);
__device__ void gpuShiftRows(int *);
__device__ int gpumul(int,int);
__device__ int gpudataget(void*, int);
__device__ void gpuMixColumns();
__device__ void gpuAddRoundKey(int *, int *, int);
__device__ void PrintPlainText(int *);
__device__ void gpuMixColumns(int *);
__device__ void ComputeTBoxes(int *,int *,int *,int *,int *);
__device__ unsigned long ConCat(unsigned char, unsigned char, unsigned char, unsigned char);
__device__ unsigned char GFMult(unsigned char , unsigned char );
__device__ void TBoxLUP(int *, int *, int *, int *);
__device__ void gpuCipher(int *, int *,int *);
__device__ void gpuCipher2(int *state, int *rkey, int *sbox, int *Tbox0, int *Tbox1, int *Tbox2, int *Tbox3);
__global__ void device_aes_encrypt(unsigned char *pt, int *rkey, unsigned char *ct, long int size){
    __shared__ int shareSbox[256];
    __shared__ int TBox0[256];
    __shared__ int TBox1[256];
    __shared__ int TBox2[256];
    __shared__ int TBox3[256];

    int gpuSbox[256] = {
         0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
         0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
         0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
         0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
         0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
         0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
         0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
         0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
         0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
         0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
         0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
         0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
         0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
         0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
         0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
         0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
    };
    memcpy(shareSbox, gpuSbox, sizeof(int) * 256);
    __syncthreads();
    ComputeTBoxes(shareSbox, TBox0, TBox1, TBox2, TBox3);
    __syncthreads();
    //This kernel executes AES encryption on a GPU.
    //Please modify this kernel!!

    int data[NBb];
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    memcpy(data, pt+16*thread_id, NBb); //With NB, 16 bytes are defined as 4 words.

    gpuCipher2(data, rkey, shareSbox, TBox0, TBox1, TBox2, TBox3);
    memcpy(ct+16*thread_id, data, NBb);
}


__device__ void gpuSubBytes(int *state, int *gpuSbox){
    int i, j;
    unsigned char *cb=(unsigned char*)state;
#pragma unroll
    for(i=0; i<NBb; i+=4){
        for(j=0; j<4; j++){
            cb[i+j] = gpuSbox[cb[i+j]];
        }
    }
}

// __device__ void gpuShiftRows(int *state){
//   int i, j, i4;
//   unsigned char *cb = (unsigned char*)state;
//   unsigned char cw[NBb];
//   memcpy(cw, cb, sizeof(cw));
//
//   for(i = 0;i < NB; i+=4){
//     i4 = i*4;
// #pragma unroll
//     for(j = 1; j < 4; j++){
//       cw[i4+j+0*4] = cb[i4+j+((j+0)&3)*4];
//       cw[i4+j+1*4] = cb[i4+j+((j+1)&3)*4];
//       cw[i4+j+2*4] = cb[i4+j+((j+2)&3)*4];
//       cw[i4+j+3*4] = cb[i4+j+((j+3)&3)*4];
//     }
//   }
//   memcpy(cb,cw,sizeof(cw));
// }
__device__ void gpuShiftRows(int *state){
  unsigned char *cb = (unsigned char*)state;
  unsigned char cw[NBb];
  memcpy(cw, cb, sizeof(cw));
  cw[1] = cb[5];
  cw[5] = cb[9];
  cw[9] = cb[13];
  cw[13] = cb[1];
  cw[2] = cb[10];
  cw[6] = cb[14];
  cw[10] = cb[2];
  cw[14] = cb[6];
  cw[3] = cb[15];
  cw[7] = cb[3];
  cw[11] = cb[7];
  cw[15] = cb[11];

  memcpy(cb,cw,sizeof(cw));
}

__device__ int gpumul(int dt,int n){
  int i, x = 0;
#pragma unroll
  for(i = 8; i > 0; i >>= 1)
    {
      x <<= 1;
      if(x & 0x100)
        x = (x ^ 0x1b) & 0xff;
      if((n & i))
        x ^= dt;
    }
  return(x);
}

__device__ int gpudataget(void* data, int n){
  return(((unsigned char*)data)[n]);
}

__device__ void gpuMixColumns(int *state){
  int i, i4, x;
  for(i = 0; i< NB; i++){
    i4 = i*4;
    x  =  gpumul(gpudataget(state,i4+0),2) ^
          gpumul(gpudataget(state,i4+1),3) ^
          gpumul(gpudataget(state,i4+2),1) ^
          gpumul(gpudataget(state,i4+3),1);
    x |= (gpumul(gpudataget(state,i4+1),2) ^
          gpumul(gpudataget(state,i4+2),3) ^
          gpumul(gpudataget(state,i4+3),1) ^
          gpumul(gpudataget(state,i4+0),1)) << 8;
    x |= (gpumul(gpudataget(state,i4+2),2) ^
          gpumul(gpudataget(state,i4+3),3) ^
          gpumul(gpudataget(state,i4+0),1) ^
          gpumul(gpudataget(state,i4+1),1)) << 16;
    x |= (gpumul(gpudataget(state,i4+3),2) ^
          gpumul(gpudataget(state,i4+0),3) ^
          gpumul(gpudataget(state,i4+1),1) ^
          gpumul(gpudataget(state,i4+2),1)) << 24;
    state[i] = x;
  }
}

__device__ void gpuAddRoundKey(int *state, int *w, int n){
    int i;
#pragma unroll
    for(i = 0; i < NB; i++) {
        state[i] ^= w[i + NB * n];
    }
}
__device__ void PrintPlainText(int *state){
  int i;
  unsigned char *cdt = (unsigned char *)state;
  for (i = 0; i < 16; i++) {
    printf("%02x", cdt[i]);
  }
  printf("\n");
}
// __device__ void gpudatadump(const char *c, void *dt, int len){
//   int i;
//   unsigned char *cdt = (unsigned char *)dt;
//   printf("%s", c);
//   for(i = 0; i < len*4;i++){
//     printf("%02x", cdt[i]);
//   }
//   printf("\n");
// }

// concatenate four byte to a dword
__device__ unsigned long ConCat(unsigned char b0, unsigned char b1, unsigned char b2, unsigned char b3){
	unsigned long dwDword = 0;
	dwDword += b0;
	dwDword = (dwDword << 8);
	dwDword += b1;
	dwDword = (dwDword << 8);
	dwDword += b2;
	dwDword = (dwDword << 8);
	dwDword += b3;
	return dwDword;
}

// multiply in GF 2^8 and reduce by AES polynom if necessary
__device__ unsigned char GFMult(unsigned char bFac1, unsigned char bFac2) {
	unsigned char p = 0;
	unsigned char counter;
	unsigned char hi_bit_set;
	for(counter = 0; counter < 8; counter++) {
		if((bFac2 & 1) == 1)
			p ^= bFac1;
		hi_bit_set = (bFac1 & 0x80);
		bFac1 <<= 1;
		if(hi_bit_set == 0x80)
			bFac1 ^= 0x1b;
		bFac2 >>= 1;
	}
	return p;
}
__device__ void ComputeTBoxes(int *Sbox, int *TBox0, int *TBox1, int *TBox2, int *TBox3){
	for(int i = 0; i < 256; i++){
		TBox0[i] = ConCat( GFMult(Sbox[i], 02), Sbox[i], Sbox[i], GFMult(Sbox[i], 03) );
		TBox1[i] = ConCat( GFMult(Sbox[i], 03), GFMult(Sbox[i], 02), Sbox[i], Sbox[i] );
		TBox2[i] = ConCat( Sbox[i], GFMult(Sbox[i], 03), GFMult(Sbox[i], 02), Sbox[i] );
		TBox3[i] = ConCat( Sbox[i], Sbox[i], GFMult(Sbox[i], 03), GFMult(Sbox[i], 02) );
	}
}

__device__ void TBoxLUP(int *state, int *TBox0, int *TBox1, int *TBox2, int *TBox3) {

    unsigned char *cb = (unsigned char*)state;
    unsigned long e0 = TBox0[cb[0]] ^ TBox1[cb[5]] ^ TBox2[cb[10]] ^ TBox3[cb[15]];
    unsigned long e1 = TBox0[cb[4]] ^ TBox1[cb[9]] ^ TBox2[cb[14]] ^ TBox3[cb[3]];
    unsigned long e2 = TBox0[cb[8]] ^ TBox1[cb[13]] ^ TBox2[cb[2]] ^ TBox3[cb[7]];
    unsigned long e3 = TBox0[cb[12]] ^ TBox1[cb[1]] ^ TBox2[cb[6]] ^ TBox3[cb[11]];
    cb[0] = (e0 >> 24) & 0xff;
    cb[1] = (e0 >> 16) & 0xff;
    cb[2] = (e0 >> 8) & 0xff;
    cb[3] = e0 & 0xff;
    cb[4] = (e1 >> 24) & 0xff;
    cb[5] = (e1 >> 16) & 0xff;
    cb[6] = (e1 >> 8) & 0xff;
    cb[7] = e1 & 0xff;
    cb[8] = (e2 >> 24) & 0xff;
    cb[9] = (e2 >> 16) & 0xff;
    cb[10] = (e2 >> 8) & 0xff;
    cb[11] = e2 & 0xff;
    cb[12] = (e3 >> 24) & 0xff;
    cb[13] = (e3 >> 16) & 0xff;
    cb[14] = (e3 >> 8) & 0xff;
    cb[15] = e3 & 0xff;
}
__device__ void gpuCipher(int *state, int *rkey, int *sbox){
  int rnd;

  gpuAddRoundKey(state, rkey, 0);

#pragma unroll
  for(rnd = 1; rnd < NR; rnd++){
    gpuSubBytes(state, sbox);
    gpuShiftRows(state);
    gpuMixColumns(state);
    gpuAddRoundKey(state, rkey, rnd);
  }

  gpuSubBytes(state, sbox);
  gpuShiftRows(state);
  gpuAddRoundKey(state, rkey, rnd);

  //return 0;
}
__device__ void gpuCipher2(int *state, int *rkey, int *sbox, int *TBox0, int *TBox1, int *TBox2, int *TBox3){
  int rnd;

  gpuAddRoundKey(state, rkey, 0);

#pragma unroll
  for(rnd = 1; rnd <NR; rnd++){
    TBoxLUP(state, TBox0, TBox1, TBox2, TBox3);
    gpuAddRoundKey(state, rkey, rnd);
  }
  gpuSubBytes(state, sbox);
  gpuShiftRows(state);
  gpuAddRoundKey(state, rkey, rnd);
  //return 0;
}
void launch_aes_kernel(unsigned char *pt, int *rk, unsigned char *ct, long int size){

  //This function launches the AES kernel.
  //Please modify this function for AES kernel.
  //In this function, you need to allocate the device memory and so on.

  unsigned char *d_pt, *d_ct;
  int *d_rkey;
  unsigned int length = FILESIZE / STREAM_NUM;
  //unsigned int char_ptr = sizeof(unsigned char) * length;
  hipMalloc((void **)&d_pt, sizeof(unsigned char)*size);
  hipMalloc((void **)&d_rkey, sizeof(int)*44);
  hipMalloc((void **)&d_ct, sizeof(unsigned char)*size);


// TODO:Using Stream
  dim3 dim_grid(FILESIZE/16/512/STREAM_NUM,1,1), dim_block(512,1,1);
  hipStream_t streams[STREAM_NUM];
  hipMemcpy(d_rkey, rk, sizeof(int)*44, hipMemcpyHostToDevice);

  for (int i = 0; i < STREAM_NUM; i++){
      hipStreamCreate(&streams[i]);
  }

  for (int i = 0; i < STREAM_NUM; i++){
      const int curStream = i;
      int pt_d = i * length;

      hipMemcpyAsync(d_pt + pt_d, pt+ pt_d, sizeof(unsigned char)*length, hipMemcpyHostToDevice, streams[curStream]);
      device_aes_encrypt<<<dim_grid, dim_block, 0, streams[curStream]>>>(d_pt + pt_d, d_rkey, d_ct + pt_d, size);
  }

  for (int i = 0; i < STREAM_NUM; i++){
      int pt_d = i * length;
      hipMemcpyAsync(ct + pt_d, d_ct+ pt_d, sizeof(unsigned char)*length, hipMemcpyDeviceToHost, streams[i]);
  }

  for (int i = 0; i < STREAM_NUM; i++){
      hipStreamSynchronize(streams[i]);
      hipStreamDestroy(streams[i]);
  }

  //Normal Mode
  // dim3 dim_grid(FILESIZE/16/512,1,1), dim_block(512,1,1);

  // hipMalloc((void **)&d_pt, sizeof(unsigned char)*size);
  // hipMalloc((void **)&d_rkey, sizeof(int)*44);
  // hipMalloc((void **)&d_ct, sizeof(unsigned char)*size);

  // hipMemset(d_pt, 0, sizeof(unsigned char)*size);
  // hipMemcpy(d_pt, pt, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
  // hipMemcpy(d_rkey, rk, sizeof(int)*44, hipMemcpyHostToDevice);

  // device_aes_encrypt<<<dim_grid, dim_block>>>(d_pt, d_rkey, d_ct, size);

  // hipMemcpy(ct, d_ct, sizeof(unsigned char)*size, hipMemcpyDeviceToHost);

  //hipFree(d_pt);
  //hipFree(d_ct);
}
